// Reference: [1] Jason Sanders, Edward Kandrot. CUDA by Example: An Introduction to General-Purpose GPU Programming.


#include <hip/hip_runtime.h>
#include <iostream>

namespace CUDA
{
	template<class Type> class FreeFunction;

	template<class ReturnType, class ...Args>
	class FreeFunction<ReturnType(Args...)>
	{
		public:
		FreeFunction(void(*fcn)(ReturnType*, Args...)) : dataOnDevice_{nullptr}, fcn_{fcn}
		{
			allocateMemoryOnDevice();
		}
		const ReturnType& operator()(Args&& ...args)
		{
			callFunctionOnDevice(std::forward<Args>(args)...);
			retrieveResultFromDevice();
			return dataOnHost_;
		}
		~FreeFunction()
		{
			freeMemoryOnDevice();
		}
		private:
		void allocateMemoryOnDevice() { hipMalloc((void**)&dataOnDevice_, sizeof(ReturnType)); }
		void callFunctionOnDevice(Args&& ...args) { (*fcn_)<<<1, 1>>>(dataOnDevice_, std::forward<Args>(args)...); }
		void retrieveResultFromDevice() { hipMemcpy(&dataOnHost_, dataOnDevice_, sizeof(ReturnType), hipMemcpyDeviceToHost); }
		void freeMemoryOnDevice() { hipFree(dataOnDevice_); }
		private:
		ReturnType dataOnHost_;
		ReturnType* dataOnDevice_;
		void(*fcn_)(ReturnType*, Args...);
	};
}

__global__ void add(int* result, int lhs, int rhs)
{
	*result = lhs + rhs;
}

__global__ void add(double* result, double a, double b, double c, double d)
{
	*result = a + b + c + d;
}

int main()
{
	CUDA::FreeFunction<int(int, int)> addIntegers(add);	
	std::cout << addIntegers(2, 7) << "\n";
	std::cout << addIntegers(3, 10) << "\n";

	CUDA::FreeFunction<double(double, double, double, double)> addDoubles(add);
	std::cout << addDoubles(2.0, 7.0, 3.0, 10.5) << "\n";

	return 0;
}
