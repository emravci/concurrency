#include "hip/hip_runtime.h"
#include <iostream>
#include <memory>

#include "../include/ManagedMemory/Matrix.cuh"
#include "../include/ManagedMemory/Vector.cuh"

template<class Type>
__global__ void fill(ManagedMemory::Vector<Type>& array, Type value)
{
    for(std::size_t i = threadIdx.x + blockIdx.x * blockDim.x; i < array.size(); i += blockDim.x * gridDim.x) { array[i] = value; }
}

template<class Type>
__global__ void fill(ManagedMemory::Matrix<Type>& matrix, Type value)
{
    for(std::size_t i = threadIdx.y + blockIdx.y * blockDim.y; i < matrix.row(); i += blockDim.y * gridDim.y)
    {
        for(std::size_t j = threadIdx.x + blockIdx.x * blockDim.x; j < matrix.column(); j += blockDim.x * gridDim.x)
        {
            matrix(i, j) = value;
        }
    }
}

template<class Type>
__global__ void multiply(ManagedMemory::Vector<Type>& ans, const ManagedMemory::Matrix<Type>& matrix, const ManagedMemory::Vector<Type>& vector)
{   // shared vector size is assumed to be equal to threads per block
    extern __shared__ Type tiledVector[];
    std::size_t commonSize = matrix.column(); // or vector.size(); or ans.size();  
    for(std::size_t h = 0; h < (commonSize + blockDim.x - 1) / blockDim.x; ++h)
    {
        if(threadIdx.x + h * blockDim.x < commonSize) { tiledVector[threadIdx.x] = vector[threadIdx.x + h * blockDim.x]; }
        __syncthreads();
        
        for(std::size_t i = threadIdx.x + blockIdx.x * blockDim.x; i < matrix.row(); i += blockDim.x * gridDim.x)
        {
            auto value = static_cast<Type>(0);
            for(std::size_t j = 0; j + h * blockDim.x < commonSize && j < blockDim.x; ++j) { value += matrix(i, j + h * blockDim.x) * tiledVector[j]; }
            ans[i] += value;
        }
        __syncthreads();
    }
}

template<class Type>
bool checkResult(const ManagedMemory::Vector<Type>& vector, Type value)
{
    for(std::size_t i = 0; i < vector.size(); ++i) { if(vector[i] != value) { return false; } }
    return true;
}

int main()
{
    using MatrixType = ManagedMemory::Matrix<double>;
    using VectorType = ManagedMemory::Vector<double>;
    constexpr std::size_t row = 1024 * 2;
    constexpr std::size_t col = 1024 * 4;
    auto pMatrix = std::make_unique<MatrixType>(row, col);
    {   // fill with ones
        dim3 threadsPerBlock(32, 32);
        dim3 blocksPerGrid(32, 32);
        fill<<<blocksPerGrid, threadsPerBlock>>>(*pMatrix, 1.0);
    }
    auto pVector = std::make_unique<VectorType>(col);
    {   // fill with ones
        dim3 threadsPerBlock(1024);
        dim3 blocksPerGrid(1);
        fill<<<blocksPerGrid, threadsPerBlock>>>(*pVector, 1.0);
    }
    auto pAnswer = std::make_unique<VectorType>(row);
    {   // matrix vector multiplication takes ?ms according to nvprof
        dim3 threadsPerBlock(1024);
        dim3 blocksPerGrid(1);
        multiply<<<blocksPerGrid ,threadsPerBlock, threadsPerBlock.x * sizeof(double)>>>(*pAnswer, *pMatrix, *pVector);
        hipDeviceSynchronize();
    }
    std::cout << std::boolalpha << checkResult(*pAnswer, static_cast<double>(col)) << "\n";
	return 0;
}
