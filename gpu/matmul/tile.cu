#include "hip/hip_runtime.h"
// Reference: [1] Wen-mei W. Hwu, David B. Kirk, Izzat El Hajj. Programming Massively Parallel Processors: A Hands-on Approach, Forth Edition.

#include <iostream>
#include <numeric>
#include <memory>
#include <chrono>
#include "include/Matrix.cuh"

struct Stride
{
    __host__ __device__ Stride(std::size_t x_, std::size_t y_ = 0, std::size_t z_ = 0) : x{x_}, y{y_}, z{z_} {}
    std::size_t x, y, z;
};

template<class Type>
__global__ void fill(UniformMemory::Matrix<Type>& matrix, Type value)
{
    const std::size_t x = threadIdx.x + blockIdx.x * blockDim.x;
    const std::size_t y = threadIdx.y + blockIdx.y * blockDim.y;
    const Stride stride{blockDim.x * gridDim.x, blockDim.y * gridDim.y};
    for(std::size_t i = x; i < matrix.row(); i += stride.x)
    {
        for(std::size_t j = y; j < matrix.column(); j += stride.y)
        {
            matrix(i, j) = value;
        }
    }
}

// tileWidth is assumed to be equal to blockDims 
constexpr std::size_t tileWidth = 32;

template<class Type>
__global__ void multiply(UniformMemory::Matrix<Type>& answer, const UniformMemory::Matrix<Type>& lhs, const UniformMemory::Matrix<Type>& rhs)
{
    __shared__ Type leftTile[tileWidth][tileWidth];
    __shared__ Type rightTile[tileWidth][tileWidth];
    
    const std::size_t x = threadIdx.x + blockIdx.x * blockDim.x;
    const std::size_t y = threadIdx.y + blockIdx.y * blockDim.y;
    const Stride stride{blockDim.x * gridDim.x, blockDim.y * gridDim.y};

    const std::size_t commonSize = lhs.column();
    for(std::size_t i = x; i < lhs.row(); i += stride.x)
    {
        for(std::size_t j = y; j < rhs.column(); j += stride.y)
        {
            auto value = static_cast<Type>(0);
            for(std::size_t h = 0; h < commonSize / tileWidth; ++h)
            {
                if(threadIdx.y + h * tileWidth < commonSize) { leftTile[threadIdx.x][threadIdx.y] = lhs(i, threadIdx.y + h * tileWidth); }
                else { leftTile[threadIdx.x][threadIdx.y] = static_cast<Type>(0); }
                if(threadIdx.x + h * tileWidth < commonSize) { rightTile[threadIdx.x][threadIdx.y] = rhs(threadIdx.x + h * tileWidth, j); }
                else { rightTile[threadIdx.x][threadIdx.y] = static_cast<Type>(0); }
                __syncthreads();

                for(std::size_t k = 0; k < tileWidth; ++k)
                {
                    value += leftTile[threadIdx.x][k] * rightTile[k][threadIdx.y];
                }
                __syncthreads();
            }
            answer(i, j) = value;
        }
    }
}

template<class Type>
bool checkResult(const UniformMemory::Matrix<Type>& matrix, const Type value)
{
    for(std::size_t i = 0; i < matrix.row(); ++i)
    {
        for(std::size_t j = 0; j < matrix.column(); ++j)
        {
            if(matrix(i, j) != value) { return false; }
        }
    }
    return true;
}

int main()
{
    using MatrixType = UniformMemory::Matrix<double>;
    constexpr std::size_t lhsRow = 1024;
    constexpr std::size_t rhsCol = 1024;
    constexpr std::size_t common = 1024;
    auto pLHS = std::make_unique<MatrixType>(lhsRow, common);
    auto pRHS = std::make_unique<MatrixType>(common, rhsCol);
    {   // fill with ones
        dim3 threadsPerBlock(32, 32);
        dim3 blocksPerGrid(32, 32);
        fill<<<blocksPerGrid, threadsPerBlock>>>(*pLHS, 1.0);
        fill<<<blocksPerGrid, threadsPerBlock>>>(*pRHS, 1.0);
    }
    auto pAnswer = std::make_unique<MatrixType>(lhsRow, rhsCol);
    {   // matrix multiplication takes 95ms according to nvprof achieving at least 10% improvement
        dim3 threadsPerBlock(tileWidth, tileWidth);
        dim3 blocksPerGrid(32, 32);
        multiply<<<blocksPerGrid ,threadsPerBlock>>>(*pAnswer, *pLHS, *pRHS);
        hipDeviceSynchronize();
    }
    std::cout << std::boolalpha << checkResult(*pAnswer, static_cast<double>(common)) << "\n";
    return 0;
}
