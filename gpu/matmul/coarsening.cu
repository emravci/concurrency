#include "hip/hip_runtime.h"
// Reference: [1] Wen-mei W. Hwu, David B. Kirk, Izzat El Hajj. Programming Massively Parallel Processors: A Hands-on Approach, Forth Edition.

#include <iostream>
#include <numeric>
#include <memory>
#include <chrono>
#include "../include/Matrix.cuh"

template<class Type>
__global__ void fill(UnifiedMemory::Matrix<Type>& matrix, Type value)
{
    for(std::size_t i = threadIdx.y + blockIdx.y * blockDim.y; i < matrix.row(); i += blockDim.y * gridDim.y)
    {
        for(std::size_t j = threadIdx.x + blockIdx.x * blockDim.x; j < matrix.column(); j += blockDim.x * gridDim.x)
        {
            matrix(i, j) = value;
        }
    }
}

// tileWidth is assumed to be equal to blockDims 
constexpr std::size_t tileWidth = 32;
constexpr std::size_t coarseFactor = 8;

template<class Type>
__global__ void multiply(UnifiedMemory::Matrix<Type>& answer, const UnifiedMemory::Matrix<Type>& lhs, const UnifiedMemory::Matrix<Type>& rhs)
{
    __shared__ Type leftTile[tileWidth][tileWidth];
    __shared__ Type rightTile[tileWidth][tileWidth];

    const auto zero = static_cast<Type>(0);
    const std::size_t commonSize = lhs.column();
    Type values[coarseFactor];
    for(std::size_t i = threadIdx.y + blockIdx.y * blockDim.y; i < lhs.row(); i += blockDim.y * gridDim.y)
    {
        for(std::size_t j = threadIdx.x + blockIdx.x * blockDim.x * coarseFactor; j < rhs.column(); j += blockDim.x * gridDim.x * coarseFactor)
        {
            for(auto& value : values) { value = zero; }
            for(std::size_t h = 0; h < (commonSize + tileWidth - 1) / tileWidth; ++h)
            {
                if(threadIdx.x + h * tileWidth < commonSize) { leftTile[threadIdx.y][threadIdx.x] = lhs(i, threadIdx.x + h * tileWidth); }
                else { leftTile[threadIdx.y][threadIdx.x] = zero; }
                
                for(std::size_t c = 0; c < coarseFactor; ++c)
                {
                    if(threadIdx.y + h * tileWidth < commonSize && j + c * tileWidth < rhs.column())
                    { 
                        rightTile[threadIdx.y][threadIdx.x] = rhs(threadIdx.y + h * tileWidth, j + c * tileWidth); 
                    }
                    else { rightTile[threadIdx.y][threadIdx.x] = zero; }
                    __syncthreads();

                    for(std::size_t k = 0; k < tileWidth; ++k) { values[c] += leftTile[threadIdx.y][k] * rightTile[k][threadIdx.x]; }
                    __syncthreads();
                }
            }
            for(std::size_t c = 0; c < coarseFactor; ++c) { answer(i, j + c * tileWidth) = values[c]; }
        }
    }
}

template<class Type>
bool checkResult(const UnifiedMemory::Matrix<Type>& matrix, const Type value)
{
    for(std::size_t i = 0; i < matrix.row(); ++i)
    {
        for(std::size_t j = 0; j < matrix.column(); ++j)
        {
            if(matrix(i, j) != value) { return false; }
        }
    }
    return true;
}

int main()
{
    using MatrixType = UnifiedMemory::Matrix<double>;
    constexpr std::size_t lhsRow = 1024;
    constexpr std::size_t rhsCol = 1024;
    constexpr std::size_t common = 1024;
    auto pLHS = std::make_unique<MatrixType>(lhsRow, common);
    auto pRHS = std::make_unique<MatrixType>(common, rhsCol);
    {   // fill with ones
        dim3 threadsPerBlock(32, 32);
        dim3 blocksPerGrid(32, 32);
        fill<<<blocksPerGrid, threadsPerBlock>>>(*pLHS, 1.0);
        fill<<<blocksPerGrid, threadsPerBlock>>>(*pRHS, 1.0);
    }
    auto pAnswer = std::make_unique<MatrixType>(lhsRow, rhsCol);
    {   // matrix multiplication takes 41ms according to nvprof 
        // achieving at least 40% improvement on tile 55% improvement on naive algorithm
        dim3 threadsPerBlock(tileWidth, tileWidth);
        dim3 blocksPerGrid(32 / coarseFactor, 32);
        multiply<<<blocksPerGrid ,threadsPerBlock>>>(*pAnswer, *pLHS, *pRHS);
        hipDeviceSynchronize();
    }
    std::cout << std::boolalpha << checkResult(*pAnswer, static_cast<double>(common)) << "\n";
    return 0;
}
